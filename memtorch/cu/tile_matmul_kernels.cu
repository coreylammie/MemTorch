#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "utils.cuh"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <iostream>
#include <limits>
#include <math.h>
#include <torch/types.h>

#include <Eigen/Core>
#include <Eigen/SparseCore>
#include <Eigen/SparseQR>

#include "quantize.cuh"
// #include "solve_passive.cuh"
// #include "solve_sparse_linear.h"
#include "solve_passive_kernels.cuh"

using namespace torch::indexing;

__global__ void tile_matmul_kernel(
    float *mat_a_tiles_accessor,
    torch::PackedTensorAccessor32<float, 1> mat_a_tiles_map_accessor,
    int64_t *mat_a_tiles_shape, float *mat_b_tiles_accessor,
    torch::PackedTensorAccessor32<float, 2> mat_b_tiles_map_accessor,
    int64_t *mat_b_tiles_shape, int mat_b_shape_back, int limit_i, int limit_j,
    int limit_k, float *result) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  int k = threadIdx.z + blockIdx.z * blockDim.z;
  if (i < limit_i && j < limit_j && k < limit_k) {
    Eigen::Map<Eigen::MatrixXf> tile_a(
        &mat_a_tiles_accessor[transform_3d_index(mat_a_tiles_map_accessor[k], i,
                                                 0, mat_a_tiles_shape[1],
                                                 mat_a_tiles_shape[2])],
        1, mat_a_tiles_shape[2]);
    Eigen::Map<Eigen::MatrixXf, Eigen::RowMajor,
               Eigen::Stride<1, Eigen::Dynamic>>
        tile_b(&mat_b_tiles_accessor[transform_3d_index(
                   mat_b_tiles_map_accessor[k][j], 0, 0, mat_b_tiles_shape[1],
                   mat_b_tiles_shape[2])],
               mat_b_tiles_shape[1], mat_b_tiles_shape[2],
               Eigen::Stride<1, Eigen::Dynamic>(1, mat_b_tiles_shape[2]));
    Eigen::VectorXf partial_sum = (tile_a * tile_b).transpose();
    for (int ii = 0; ii < partial_sum.size(); ii++) {
      result[transform_2d_index(i, j * mat_b_tiles_shape[2] + ii,
                                mat_b_shape_back)] += partial_sum[ii];
    }
    free(&partial_sum);
  }
}

// __global__ void tile_matmul_kernel_A(
//     float *mat_a_tiles_accessor,
//     torch::PackedTensorAccessor32<float, 1> mat_a_tiles_map_accessor,
//     int64_t *mat_a_tiles_shape, float *mat_b_tiles_accessor,
//     torch::PackedTensorAccessor32<float, 2> mat_b_tiles_map_accessor,
//     int64_t *mat_b_tiles_shape, int mat_b_shape_back,
//     int *ABCD_matrix_indices_x, int *ABCD_matrix_indices_y,
//     double *ABCD_matrix_values, int *ABCD_matrix_compressed_rows,
//     int *ABCD_matrix_compressed_columns, double *ABCD_matrix_compressed_values,
//     double *E_matrix, float source_resistance, float line_resistance,
//     int limit_i, int limit_j, int limit_k) {
//   int i = threadIdx.x + blockIdx.x * blockDim.x;
//   int j = threadIdx.y + blockIdx.y * blockDim.y;
//   int k = threadIdx.z + blockIdx.z * blockDim.z;
//   if (i < limit_i && j < limit_j && k < limit_k) {
//     Eigen::Map<Eigen::VectorXf> tile_a(
//         &mat_a_tiles_accessor[transform_3d_index(mat_a_tiles_map_accessor[k], i,
//                                                  0, mat_a_tiles_shape[1],
//                                                  mat_a_tiles_shape[2])],
//         mat_a_tiles_shape[1]);
//     Eigen::Map<Eigen::MatrixXf, Eigen::RowMajor,
//                Eigen::Stride<1, Eigen::Dynamic>>
//         tile_b(&mat_b_tiles_accessor[transform_3d_index(
//                    mat_b_tiles_map_accessor[k][j], 0, 0, mat_b_tiles_shape[1],
//                    mat_b_tiles_shape[2])],
//                mat_b_tiles_shape[1], mat_b_tiles_shape[2],
//                Eigen::Stride<1, Eigen::Dynamic>(1, mat_b_tiles_shape[2]));
//     int m = (int)mat_b_tiles_shape[1];
//     int n = (int)mat_b_tiles_shape[2];
//     int nonzero_elements = 8 * m * n - 2 * m - 2 * n;
//     int kernel_index = transform_3d_index(i, j, k, limit_j, limit_k);
//     construct_ABCD_E(
//         tile_b, tile_a, Eigen::VectorXf::Zero(n), source_resistance,
//         line_resistance,
//         &ABCD_matrix_indices_x[kernel_index * nonzero_elements],
//         &ABCD_matrix_indices_y[kernel_index * nonzero_elements],
//         &ABCD_matrix_values[kernel_index * nonzero_elements],
//         &ABCD_matrix_compressed_rows[kernel_index * nonzero_elements],
//         &ABCD_matrix_compressed_columns[kernel_index * (2 * m * n)],
//         &ABCD_matrix_compressed_values[kernel_index * nonzero_elements],
//         &E_matrix[kernel_index * (2 * m * n)]);
//   }
// }

// __global__ void tile_matmul_kernel_B(
//     double *E_matrix, float *mat_b_tiles_accessor,
//     torch::PackedTensorAccessor32<float, 2> mat_b_tiles_map_accessor,
//     int64_t *mat_b_tiles_shape, int mat_b_shape_back, int m, int n, int limit_i,
//     int limit_j, int limit_k, float *result) {
//   int i = threadIdx.x + blockIdx.x * blockDim.x;
//   int j = threadIdx.y + blockIdx.y * blockDim.y;
//   int k = threadIdx.z + blockIdx.z * blockDim.z;
//   if (i < limit_i && j < limit_j && k < limit_k) {
//     int kernel_index = transform_3d_index(i, j, k, limit_j, limit_k);
//     Eigen::Map<Eigen::MatrixXf, Eigen::RowMajor,
//                Eigen::Stride<1, Eigen::Dynamic>>
//         tile_b(&mat_b_tiles_accessor[transform_3d_index(
//                    mat_b_tiles_map_accessor[k][j], 0, 0, mat_b_tiles_shape[1],
//                    mat_b_tiles_shape[2])],
//                mat_b_tiles_shape[1], mat_b_tiles_shape[2],
//                Eigen::Stride<1, Eigen::Dynamic>(1, mat_b_tiles_shape[2]));
//     Eigen::MatrixXf I_applied_tensor = Eigen::MatrixXf::Zero(m, n);
//     for (int ii = 0; ii < m; ii++) {
//       for (int jj = 0; jj < n; jj++) {
//         I_applied_tensor(ii, jj) =
//             ((float)E_matrix[kernel_index * (2 * m * n) + n * ii + jj] -
//              (float)
//                  E_matrix[kernel_index * (2 * m * n) + m * n + n * ii + jj]) *
//             tile_b(ii, jj);
//       }
//     }
//     Eigen::VectorXf I_tensor = I_applied_tensor.colwise().sum();
//     for (int ii = 0; ii < n; ii++) {
//       result[transform_2d_index(i, j * mat_b_tiles_shape[2] + ii,
//                                 mat_b_shape_back)] += I_tensor[ii];
//     }
//   }
// }

__global__ void tile_matmul_kernel(
    float *mat_a_tiles_accessor,
    torch::PackedTensorAccessor32<float, 1> mat_a_tiles_map_accessor,
    int64_t *mat_a_tiles_shape, float *mat_b_tiles_accessor,
    torch::PackedTensorAccessor32<float, 2> mat_b_tiles_map_accessor,
    int64_t *mat_b_tiles_shape, int mat_b_shape_back, int ADC_resolution,
    float overflow_rate, int quant_method, int limit_i, int limit_j,
    int limit_k, float *result) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  int k = threadIdx.z + blockIdx.z * blockDim.z;
  if (i < limit_i && j < limit_j && k < limit_k) {
    Eigen::Map<Eigen::MatrixXf> tile_a(
        &mat_a_tiles_accessor[transform_3d_index(mat_a_tiles_map_accessor[k], i,
                                                 0, mat_a_tiles_shape[1],
                                                 mat_a_tiles_shape[2])],
        1, mat_a_tiles_shape[2]);

    Eigen::Map<Eigen::MatrixXf, Eigen::RowMajor,
               Eigen::Stride<1, Eigen::Dynamic>>
        tile_b(&mat_b_tiles_accessor[transform_3d_index(
                   mat_b_tiles_map_accessor[k][j], 0, 0, mat_b_tiles_shape[1],
                   mat_b_tiles_shape[2])],
               mat_b_tiles_shape[1], mat_b_tiles_shape[2],
               Eigen::Stride<1, Eigen::Dynamic>(1, mat_b_tiles_shape[2]));
    Eigen::VectorXf partial_sum = (tile_a * tile_b).transpose();
    partial_sum =
        quantize(partial_sum, ADC_resolution, overflow_rate, quant_method);
#pragma omp parallel for
    for (int ii = 0; ii < partial_sum.size(); ii++) {
      result[transform_2d_index(i, j * mat_b_tiles_shape[2] + ii,
                                mat_b_shape_back)] += partial_sum[ii];
    }
    free(&partial_sum);
  }
}

__global__ void tile_matmul_kernel_B(
    double *E_matrix, float *mat_b_tiles_accessor,
    torch::PackedTensorAccessor32<float, 2> mat_b_tiles_map_accessor,
    int64_t *mat_b_tiles_shape, int mat_b_shape_back, int ADC_resolution,
    float overflow_rate, int quant_method, int m, int n, int limit_i,
    int limit_j, int limit_k, float *result) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  int k = threadIdx.z + blockIdx.z * blockDim.z;
  if (i < limit_i && j < limit_j && k < limit_k) {
    int kernel_index = transform_3d_index(i, j, k, limit_j, limit_k);
    Eigen::Map<Eigen::MatrixXf, Eigen::RowMajor,
               Eigen::Stride<1, Eigen::Dynamic>>
        tile_b(&mat_b_tiles_accessor[transform_3d_index(
                   mat_b_tiles_map_accessor[k][j], 0, 0, mat_b_tiles_shape[1],
                   mat_b_tiles_shape[2])],
               mat_b_tiles_shape[1], mat_b_tiles_shape[2],
               Eigen::Stride<1, Eigen::Dynamic>(1, mat_b_tiles_shape[2]));
    Eigen::MatrixXf I_applied_tensor = Eigen::MatrixXf::Zero(m, n);
    for (int ii = 0; ii < m; ii++) {
      for (int jj = 0; jj < n; jj++) {
        I_applied_tensor(ii, jj) =
            ((float)E_matrix[kernel_index * (2 * m * n) + n * ii + jj] -
             (float)
                 E_matrix[kernel_index * (2 * m * n) + m * n + n * ii + jj]) *
            tile_b(ii, jj);
      }
    }
    Eigen::VectorXf I_tensor = I_applied_tensor.colwise().sum();
    I_tensor = quantize(I_tensor, ADC_resolution, overflow_rate, quant_method);
    for (int ii = 0; ii < n; ii++) {
      result[transform_2d_index(i, j * mat_b_tiles_shape[2] + ii,
                                mat_b_shape_back)] += I_tensor[ii];
    }
  }
}

at::Tensor tile_matmul(at::Tensor mat_a_tiles, at::Tensor mat_a_tiles_map,
                       int mat_a_shape[2], at::Tensor mat_b_tiles,
                       at::Tensor mat_b_tiles_map, int mat_b_shape[2],
                       int ADC_resolution, float overflow_rate,
                       int quant_method, float source_resistance,
                       float line_resistance, int cuda_malloc_heap_size) {
  assert(at::cuda::is_available());
  mat_a_tiles = mat_a_tiles.to(torch::Device("cuda:0"));
  mat_a_tiles_map = mat_a_tiles_map.to(torch::Device("cuda:0"));
  mat_b_tiles = mat_b_tiles.to(torch::Device("cuda:0"));
  mat_b_tiles_map = mat_b_tiles_map.to(torch::Device("cuda:0"));
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  int *max_threads_dim = prop.maxThreadsDim;


  // hipSetDevice(0);
  // std::cout << mat_b_tiles_map << std::endl;
  // std::cout << mat_a_tiles_map << std::endl;
  // hipDeviceSynchronize();
  // hipDeviceSynchronize();
  // std::cout << "here..." << std::endl;
  // size_t free_mem;
  // size_t total_global_mem;
  // hipMemGetInfo(&free_mem, &total_global_mem);
  // std::cout << free_mem << "/ " << total_global_mem << std::endl;
  // free_mem *= 0.8;
  // size_t decrement = 1024 * 1024 * 1; // 1 MB
  // int *buf_d = NULL;
  // while(buf_d == NULL) {
  //   hipFree(buf_d);
  //   hipMalloc(&buf_d, free_mem);
  //   free_mem -= decrement;
  //   std::cout << "H" << std::endl;
  // }
  // std::cout << free_mem << std::endl;
  // std::cout << "here..." << std::endl;


  int64_t *mat_a_tiles_shape_host = (int64_t *)malloc(sizeof(int64_t) * 3);
  int64_t *mat_b_tiles_shape_host = (int64_t *)malloc(sizeof(int64_t) * 3);
  for (int i = 0; i < 3; i++) {
    mat_a_tiles_shape_host[i] = mat_a_tiles.sizes()[i];
    mat_b_tiles_shape_host[i] = mat_b_tiles.sizes()[i];
  }
  int64_t *mat_a_tiles_shape;
  int64_t *mat_b_tiles_shape;
  cudaSafeCall(hipMalloc(&mat_a_tiles_shape, sizeof(int64_t) * 3));
  cudaSafeCall(hipMalloc(&mat_b_tiles_shape, sizeof(int64_t) * 3));
  cudaSafeCall(hipMemcpy(mat_a_tiles_shape, mat_a_tiles_shape_host,
                          sizeof(int64_t) * 3, hipMemcpyHostToDevice));
  cudaSafeCall(hipMemcpy(mat_b_tiles_shape, mat_b_tiles_shape_host,
                          sizeof(int64_t) * 3, hipMemcpyHostToDevice));
  float *mat_a_tiles_accessor = mat_a_tiles.data_ptr<float>();
  float *mat_b_tiles_accessor = mat_b_tiles.data_ptr<float>();
  torch::PackedTensorAccessor32<float, 1> mat_a_tiles_map_accessor =
      mat_a_tiles_map.packed_accessor32<float, 1>();
  torch::PackedTensorAccessor32<float, 2> mat_b_tiles_map_accessor =
      mat_b_tiles_map.packed_accessor32<float, 2>();
  at::Tensor result =
      at::zeros({mat_a_shape[0], mat_b_shape[1]}, torch::device(torch::kCUDA));
// //   // hipDeviceSetLimit(hipLimitMallocHeapSize,
// //   //                    size_t(1024) * size_t(1024) *
// //   //                        size_t(cuda_malloc_heap_size));
  if (line_resistance == -1) {
    int limit_i = mat_a_tiles.sizes().end()[-2];
    int limit_j = mat_b_tiles_map.sizes()[1];
    int limit_k = mat_b_tiles_map.sizes()[0];
    dim3 grid;
    dim3 block;
    if (max_threads_dim[0] >= limit_i && max_threads_dim[1] >= limit_j &&
        max_threads_dim[2] >= limit_k) {
      // If multiple blocks are not required
      grid = {(unsigned int)limit_i, (unsigned int)limit_j,
              (unsigned int)limit_k};
      block = {1, 1, 1};
    } else {
      // If multiple blocks are required
      grid = {(unsigned int)max_threads_dim[0], (unsigned int)max_threads_dim[1],
              (unsigned int)max_threads_dim[2]};
      block = {(unsigned int)ceil_int_div(limit_i, max_threads_dim[0]),
              (unsigned int)ceil_int_div(limit_j, max_threads_dim[1]),
              (unsigned int)ceil_int_div(limit_k, max_threads_dim[2])};
    }
    if (ADC_resolution == -1) {
      tile_matmul_kernel<<<grid, block>>>(
          mat_a_tiles_accessor, mat_a_tiles_map_accessor, mat_a_tiles_shape,
          mat_b_tiles_accessor, mat_b_tiles_map_accessor, mat_b_tiles_shape,
          mat_b_shape[1], limit_i, limit_j, limit_k, result.data_ptr<float>());
    } else {
      tile_matmul_kernel<<<grid, block>>>(
          mat_a_tiles_accessor, mat_a_tiles_map_accessor, mat_a_tiles_shape,
          mat_b_tiles_accessor, mat_b_tiles_map_accessor, mat_b_tiles_shape,
          mat_b_shape[1], ADC_resolution, overflow_rate, quant_method, limit_i,
          limit_j, limit_k, result.data_ptr<float>());
    }
  } else {
    // int stride = ... // TBD- stride/ limit i
    int m = mat_b_tiles_shape_host[1]; // limit j
    int n = mat_b_tiles_shape_host[2]; // limit k
    int non_zero_elements = 8 * m * n - 2 * m - 2 * n;
    int mat_a_rows = mat_a_tiles.sizes().end()[-2];
    at::Tensor partial_sum =
      at::zeros({mat_b_tiles_map.sizes()[1], mat_b_tiles_shape_host[2]}, torch::device(torch::kCUDA));
      for (int i = 0; i < mat_a_rows; i++) {
        at::Tensor mat_a_row_tiles = mat_a_tiles.index({Slice(), i, Slice()}); 
        for (int j = 0; j < mat_b_tiles_map.sizes()[0]; j++) {
          at::Tensor tile_a = mat_a_row_tiles[mat_a_tiles_map[j].item<int>()];
          for (int k = 0; k < mat_b_tiles_map.sizes()[1]; k++) {
            at::Tensor tile_b = mat_b_tiles[mat_b_tiles_map[j][k].item<int>()];
            partial_sum[k] +=
              solve_passive(tile_b, tile_a, at::zeros({tile_b.sizes()[1]}, torch::device(torch::kCUDA)),
                            source_resistance, line_resistance, true)
                  .squeeze();
          }
          result.index_put_({i, Slice()}, result.index({i, Slice()}) +
            partial_sum.flatten().index(
                {Slice(0, mat_b_shape[1])}));
          partial_sum = partial_sum.zero_();
        }
      }
      cudaSafeCall(hipDeviceSynchronize());
  }
//   cudaSafeCall(hipDeviceSynchronize());
//   cudaSafeCall(hipFree(mat_a_tiles_shape));
//   cudaSafeCall(hipFree(mat_b_tiles_shape));
//   hipStreamSynchronize(at::cuda::getCurrentCUDAStream());
  return result;
}