
#include <hip/hip_runtime.h>
void merge(float *arr, int low, int high, int mid) {
  int i, j, k;
  float c[50];
  i = low;
  k = low;
  j = mid + 1;
  while (i <= mid && j <= high) {
    if (arr[i] > arr[j]) {
      c[k] = arr[i];
      k++;
      i++;
    } else {
      c[k] = arr[j];
      k++;
      j++;
    }
  }
  while (i <= mid) {
    c[k] = arr[i];
    k++;
    i++;
  }
  while (j <= high) {
    c[k] = arr[j];
    k++;
    j++;
  }
  for (i = low; i < k; i++) {
    arr[i] = c[i];
  }
}

void merge_sort(float *arr, int low, int high) {
  int mid;
  if (low < high) {
    mid = (low + high) / 2;
    merge_sort(arr, low, mid);
    merge_sort(arr, mid + 1, high);
    merge(arr, low, high, mid);
  }
}